#include "hip/hip_runtime.h"
//nvcc -std=c++17 -ccbin g++-11 battery_cuda_sim.cu -o battery_cuda_sim
#include "battery_sim.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <cstring>
#include <cerrno>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
//#include <math_functions.h>
#include <math.h>

#define SHM_NAME "/battery_sim_shm"
#define SHM_SIZE sizeof(BatterySim)

BatterySim* g_battery_sim = nullptr;

// CUDA kernel to simulate updating voltage and temperature
__global__ void update_cells(CellData* cells, int num_cells, float t) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < num_cells) {
        cells[idx].voltage += 0.001f * sinf(t + idx);
        cells[idx].temperature += 0.01f * cosf(t + idx);
    }
}

bool setup_shared_memory() {
    int fd = shm_open(SHM_NAME, O_RDWR, 0666);
    if (fd == -1) {
        std::cerr << "❌ shm_open failed: " << strerror(errno) << "\n";
        return false;
    }

    void* ptr = mmap(nullptr, SHM_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
    if (ptr == MAP_FAILED) {
        std::cerr << "❌ mmap failed: " << strerror(errno) << "\n";
        close(fd);
        return false;
    }

    g_battery_sim = static_cast<BatterySim*>(ptr);
    close(fd);
    return true;
}

int main() {
    if (!setup_shared_memory()) return 1;

    std::cout << "🧪 CUDA battery simulator started\n";

    while (true) {
        for (int r = 0; r < NUM_RACKS; ++r) {
            for (int m = 0; m < NUM_MODULES; ++m) {
                CellData* dev_cells;
                auto& module = g_battery_sim->racks[r].modules[m];

                hipMalloc(&dev_cells, sizeof(CellData) * NUM_CELLS);
                hipMemcpy(dev_cells, module.cells, sizeof(CellData) * NUM_CELLS, hipMemcpyHostToDevice);

                float t = static_cast<float>(time(nullptr));
                update_cells<<<(NUM_CELLS + 31)/32, 32>>>(dev_cells, NUM_CELLS, t);
                hipDeviceSynchronize();

                hipMemcpy(module.cells, dev_cells, sizeof(CellData) * NUM_CELLS, hipMemcpyDeviceToHost);
                hipFree(dev_cells);
            }
        }

        std::this_thread::sleep_for(std::chrono::seconds(1));
    }

    return 0;
}